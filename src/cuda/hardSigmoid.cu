#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "util.h"

__global__ void hardSigmoidKernel(float* input,float* output,int N) {
    // int row = blockIdx.y * blockDim.y + threadIdx.y;
    int idx= blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        output[idx]=max(0.0f, min(1.0f,(input[idx]+1.0f)/2.0f));;
        
    }
}

void hardSigmoidCUDA(float* h_A, float* h_B, int N) {
    // 分配设备内存
    float *d_A, *d_B;
    hipMalloc((void**)&d_A, N *sizeof(float));
    hipMalloc((void**)&d_B,  N * sizeof(float));

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, N* sizeof(float), hipMemcpyHostToDevice);

    // 定义线程块和网格大小
    dim3 threadsPerBlock(16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // 启动核函数
    hardSigmoidKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B,N);

    // 将结果从设备复制回主机
    hipMemcpy(h_B, d_B,  N * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
}

void hardSigmoid(const std::vector<float>& input, std::vector<float>& output) {
    
    for (size_t i = 0; i < input.size(); ++i) {
        output[i] = std::max(0.0f, std::min(1.0f,(input[i]+1.0f)/2.0f));
    }
    
}

int main() {
    int n=1024*1024;
    std::vector<float> input(n,0);
    for(int i=0;i<n;++i){
        input[i]=i;
    }

    std::vector<float> output1(n,0);   
    std::vector<float> output2(n,0);   

    auto time1=measureExecutionTime(hardSigmoid,input, output1);
    auto time2=measureExecutionTime(hardSigmoidCUDA,input.data(), output2.data(),n);


    std::cout << "Elapsed time: " << time1 << " seconds" << std::endl;     
    std::cout << "Elapsed time: " << time2 << " seconds" << std::endl;  
    check(output1.data(), output2.data(), n);

    return 0;
}