#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "util.h"


__global__ void crossenKernel(float* A,float* B,float* output,int N) {
    // int row = blockIdx.y * blockDim.y + threadIdx.y;
    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    int tid=threadIdx.x;
    extern __shared__ float sharedData[];

    if (idx < N) {
        sharedData[tid]=A[idx]*logf(B[idx])+(1-A[idx])*logf(1-B[idx]);
    }

    // 使用归约算法进行累加
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, -sharedData[0]/N);
    }
}

void crossEntropyCUDA(float* h_A, float* h_B, float& output, int N) {
    // 分配设备内存
    float *d_A, *d_B, *d_output;
    hipMalloc((void**)&d_A, N *sizeof(float));
    hipMalloc((void**)&d_B,  N * sizeof(float));
    hipMalloc((void**)&d_output,  sizeof(float));

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, N* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N* sizeof(float), hipMemcpyHostToDevice);

    // 定义线程块和网格大小
    dim3 threadsPerBlock(16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // 启动核函数
    crossenKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B,d_output,N);

    // 将结果从设备复制回主机
    hipMemcpy(&output, d_output,  sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
}

void crossEntropy(const float* y_true, const float* y_pred,float& output,const int n) {

    float sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        if (y_pred[i] < 0 || y_pred[i] > 1) {
            throw std::invalid_argument("Predicted probabilities must be in the range (0, 1).");
        }
        sum += y_true[i] * std::log(y_pred[i]) + (1 - y_true[i]) * std::log(1 - y_pred[i]);
    }
    output= -sum / n;
}


int main() {
    int n=2048;
    std::vector<float> y_true(n);
    std::vector<float> y_pred(n);
    float output1;
    float output2;
    for(int i=0;i<n;++i){
        y_true[i]=0.1;
        y_pred[i]=0.2;
    }

    auto time1=measureExecutionTime(crossEntropy,y_true.data(), y_pred.data(), output1,n);
    auto time2=measureExecutionTime(crossEntropyCUDA,y_true.data(), y_pred.data(), output2,n);
    std::cout << "Cross-Entropy: " << output1 <<" time:" <<time1<< std::endl;
    std::cout << "Cross-Entropy: " << output2 <<" time:" <<time2<< std::endl;
    

    return 0;
}