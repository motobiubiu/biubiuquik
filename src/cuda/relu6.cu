#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "util.h"

__global__ void relu6Kernel(float* A,float* output,int N) {
    // int row = blockIdx.y * blockDim.y + threadIdx.y;
    int idx= blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        float cur;
        cur=A[idx]>0?A[idx]:0;
        output[idx]=cur<6?cur:6;
    }
}

void relu6CUDA(float* h_A, float* h_B, int N) {
    // 分配设备内存
    float *d_A, *d_B;
    hipMalloc((void**)&d_A, N *sizeof(float));
    hipMalloc((void**)&d_B,  N * sizeof(float));

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, N* sizeof(float), hipMemcpyHostToDevice);

    // 定义线程块和网格大小
    dim3 threadsPerBlock(16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // 启动核函数
    relu6Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B,N);

    // 将结果从设备复制回主机
    hipMemcpy(h_B, d_B,  N * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
}

void relu6(const std::vector<float>& input, std::vector<float>& output) {
    
    for (size_t i = 0; i < input.size(); ++i) {
        output[i] = std::min(std::max(0.0f, input[i]),6.0f);
    }
    
}

int main() {

    int n=1024*1024;
    std::vector<float> input(n,0);
    for(int i=0;i<n;++i){
        input[i]=i;
    }

    std::vector<float> output1(n,0);   
    std::vector<float> output2(n,0);   

    auto time1=measureExecutionTime(relu6,input, output1);
    auto time2=measureExecutionTime(relu6CUDA,input.data(), output2.data(),n);

    std::cout << "Elapsed time: " << time1 << " seconds" << std::endl;     
    std::cout << "Elapsed time: " << time2 << " seconds" << std::endl;       

    check(output1.data(), output2.data(), n);


}