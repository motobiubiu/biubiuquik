#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "util.h"

__global__ void hardWishKernel(float* input,float* output,int N) {
    // int row = blockIdx.y * blockDim.y + threadIdx.y;
    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<N){
    if(input[idx]<=-3.0f){
            output[idx]=0;
        }else if(input[idx]>=3){
            output[idx]=input[idx];
        }else{
            output[idx]=input[idx]*(input[idx]+3.0f)/6.0f;
        }
    }
}

void hardWishCUDA(float* h_A, float* h_B, int N) {
    // 分配设备内存
    float *d_A, *d_B;
    hipMalloc((void**)&d_A, N *sizeof(float));
    hipMalloc((void**)&d_B,  N * sizeof(float));

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, N* sizeof(float), hipMemcpyHostToDevice);

    // 定义线程块和网格大小
    dim3 threadsPerBlock(16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // 启动核函数
    hardWishKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B,N);

    // 将结果从设备复制回主机
    hipMemcpy(h_B, d_B,  N * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
}

void hardWish(const std::vector<float>& input, std::vector<float>& output) {
    
    for (size_t i = 0; i < input.size(); ++i) {
        if(input[i]<=-3.0f){
            output[i]=0;
        }else if(input[i]>=3.0f){
            output[i]=input[i];
        }else{
            output[i]=input[i]*(input[i]+3.0f)/6.0f;
        }
    }
    
}

int main() {

    int n=1024*1024;
    std::vector<float> input(n,0);
    for(int i=0;i<n;++i){
        input[i]=i;
    }

    std::vector<float> output1(n,0);   
    std::vector<float> output2(n,0);   

    auto time1=measureExecutionTime(hardWish,input, output1);
    auto time2=measureExecutionTime(hardWishCUDA,input.data(), output2.data(),n);

    std::cout << "Elapsed time: " << time1 << " seconds" << std::endl;     
    std::cout << "Elapsed time: " << time2 << " seconds" << std::endl;

    check(output1.data(), output2.data(), n); 

    return 0;
}